#include "hip/hip_runtime.h"
/*
 *  Copyright 2020 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/printer_multiplex.cuh>

#include <iostream>

namespace nvbench
{

printer_multiplex::printer_multiplex()
    : printer_base(std::cerr) // Nothing should write to this.
{}

void printer_multiplex::do_print_device_info()
{
  for (auto &format_ptr : m_printers)
  {
    format_ptr->print_device_info();
  }
}

void printer_multiplex::do_print_log_preamble()
{
  for (auto &format_ptr : m_printers)
  {
    format_ptr->print_log_preamble();
  }
}

void printer_multiplex::do_print_log_epilogue()
{
  for (auto &format_ptr : m_printers)
  {
    format_ptr->print_log_epilogue();
  }
}

void printer_multiplex::do_log(nvbench::log_level level, const std::string &str)
{
  for (auto &format_ptr : m_printers)
  {
    format_ptr->log(level, str);
  }
}

void printer_multiplex::do_log_run_state(const nvbench::state &exec_state)
{
  for (auto &format_ptr : m_printers)
  {
    format_ptr->log_run_state(exec_state);
  }
}

void printer_multiplex::do_print_benchmark_list(const benchmark_vector &benches)
{
  for (auto &format_ptr : m_printers)
  {
    format_ptr->print_benchmark_list(benches);
  }
}

void printer_multiplex::do_print_benchmark_results(
  const benchmark_vector &benches)
{
  for (auto &format_ptr : m_printers)
  {
    format_ptr->print_benchmark_results(benches);
  }
}

} // namespace nvbench
